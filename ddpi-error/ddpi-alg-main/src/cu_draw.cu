#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "cu_helper.cuh"
#include "cu_draw.cuh"
#include <algorithm>

__global__ void cuDrawCircleKernel(
        unsigned char *dst, int step,
        int width, int height,
        float xc, float yc, float r,
        unsigned char R, unsigned char G, unsigned char B,
        float lw) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx > width) | (ty > height)) {
        return;
    }

    float dx = ((float)tx - xc);
    float dy = ((float)ty - yc);
    float d = sqrtf(dx * dx + dy * dy);
    if ((d >= r) && (d < (r + lw))) {
        int idx = 3 * (tx + ty * step);
        dst[idx] = R;
        dst[idx + 1] = G;
        dst[idx + 2] = B;
    }
}

__global__ void cuDrawRectKernel(
        unsigned char *dst, int step,
        int width, int height,
        int x, int y,
        int xend, int yend,
        unsigned char R, unsigned char G, unsigned char B,
        unsigned char lw) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (((tx >= x) & (tx < xend) &
         (ty >= y) & (ty < yend)) &
        ((tx < x + lw) | (tx >= xend -lw) |
         (ty < y + lw) | (ty >= yend -lw))) {
        int idx = 3 * (tx + ty * step);
            dst[idx] = R;
            dst[idx + 1] = G;
            dst[idx + 2] = B;
    }
}

int cuDrawCircle(
        unsigned char *dst, int step,
        int width, int height,
        float xc, float yc, float r,
        unsigned char R, unsigned char G, unsigned char B,
        float lw,
		hipStream_t& ctx) {
//    if ((xc < 0) | (xc >= width) | (yc < 0) | (yc >= width) | (r < 0)) {
//        return hipSuccess;
//    }

	dim3 dimBlock(32, 32);
	dim3 dimGrid((width - 1 + 32) / 32,
				 (height - 1 + 32) / 32);
    cuDrawCircleKernel<<<dimGrid, dimBlock, 0, ctx>>>(
            dst, step, width, height, xc, yc, r, R, G, B, lw);

	return hipSuccess;
}


int cuDrawRect(
        unsigned char *dst, int step,
        int width, int height,
        int xc, int yc, int w, int h,
        unsigned char R, unsigned char G, unsigned char B,
        unsigned char lw,
        hipStream_t& ctx) {

    dim3 dimBlock(32, 32);
    dim3 dimGrid((width - 1 + 32) / 32,
                 (height - 1 + 32) / 32);


    int x = xc - w/2;
    x = std::max(0, x);
    int xend = x + w;
    xend = std::min(xend, width);
    int y = yc - w/2;
    y = std::max(0, y);
    int yend = y + h;
    xend = std::min(yend, height);

    cuDrawRectKernel<<<dimGrid, dimBlock, 0, ctx>>>(
            dst, step, width, height, x, y, x + w, y + h, R, G, B, lw);

    return hipSuccess;
}
