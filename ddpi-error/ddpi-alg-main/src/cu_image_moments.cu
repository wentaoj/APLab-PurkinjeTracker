#include "hip/hip_runtime.h"
#include "cu_helper.cuh"
#include "cu_image_moments.cuh"

template<typename InputT, typename OutputT>
__device__ void imageMoments(
        InputT* pSrc,
        OutputT& m1, OutputT& m2, OutputT& m3,
        int idx,
        int width, int height, int step,
        OutputT threshold) {
    OutputT x = static_cast<OutputT>(idx % step);
    OutputT y = static_cast<OutputT>(idx / step);
    if ((x < static_cast<OutputT>(width)) & (y < static_cast<OutputT>(height))) {
        OutputT val = static_cast<OutputT>(pSrc[idx]);
        if (val < threshold) {
            val = 0;
        }
        m1 += val;
        m2 += x* val;
        m3 += y* val;
    }
}

template<typename InputT, typename OutputT>
__device__ void imageMomentsPower(
        InputT* pSrc,
        OutputT& m1, OutputT& m2, OutputT& m3,
        int idx,
        int width, int height, int step,
        OutputT power) {
    OutputT x = static_cast<OutputT>(idx % step);
    OutputT y = static_cast<OutputT>(idx / step);
    if ((x < static_cast<OutputT>(width)) & (y < static_cast<OutputT>(height))) {
        OutputT val = pow(static_cast<OutputT>(pSrc[idx]), power);
        m1 += val;
        m2 += x * val;
        m3 += y * val;
    }
}

template<class InputT, class OutputT, bool nIsPow2>
__global__ void imageMomentsKernel(
        InputT *pSrc, int step,
        OutputT *pDst,
        int width, int height,
        OutputT threshold) {

    // Shared memory for intermediate steps
    OutputT* sdata = SharedMemory<OutputT>();

    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    // Handle to tile in thread block
    cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cta);
    unsigned int n = step * height;

    unsigned int threadRank = threadIdx.x;
    unsigned int blocksize = cta.size();
    OutputT threadVal00 = 0;
    OutputT threadVal10 = 0;
    OutputT threadVal01 = 0;

    if (nIsPow2) {
        unsigned int i = blockIdx.x * blocksize * 2 + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x << 1;
        while (i < n) {
            imageMoments(
                    pSrc,
                    threadVal00, threadVal10, threadVal01,
                    i,
                    width, height, step,
                    threshold);
            // ensure we don't read out of bounds -- this is optimized away for
            // powerOf2 sized arrays
            int j = i + blocksize;
            if (j < n) {
                imageMoments(
                        pSrc,
                        threadVal00, threadVal10, threadVal01,
                        j,
                        width, height, step,
                        threshold);
            }
            i += gridSize;
        }
    }
    else {
        unsigned int i = blockIdx.x * blocksize + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x;
        while (i < n) {
            imageMoments(
                    pSrc,
                    threadVal00, threadVal10, threadVal01,
                    i,
                    width, height, step,
                    threshold);
            i += gridSize;
        }
    }

    sdata[threadRank] = threadVal00;
    sdata[threadRank + blocksize] = threadVal10;
    sdata[threadRank + 2 * blocksize] = threadVal01;
    cg::sync(cta);

    // do reduction in shared mem
    if (threadRank < 512) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 512];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 512];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 512];
    }
    cg::sync(cta);

    if (threadRank < 256) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 256];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 256];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 256];
    }
    cg::sync(cta);

    if (threadRank < 128) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 128];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 128];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 128];
    }
    cg::sync(cta);

    if (threadRank < 64) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 64];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 64];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 64];
    }
    cg::sync(cta);

    if (threadRank < 32) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 32];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 32];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 32];
    }
    cg::sync(cta);

    //reduce(threadVal00, sdata, threadRank, cta, tile, 1024);
    //reduce(threadVal10, sdata, threadRank, cta, tile, 1024);
    //reduce(threadVal01, sdata, threadRank, cta, tile, 1024);

    if (threadRank < 32) {
        // Reduce final warp using shuffle
        for (int offset = tile.size() / 2; offset > 0; offset /= 2) {
            threadVal00 += tile.shfl_down(threadVal00, offset);
            threadVal01 += tile.shfl_down(threadVal01, offset);
            threadVal10 += tile.shfl_down(threadVal10, offset);
        }
    }

    if (threadRank == 0) {
        atomicAdd(pDst, threadVal00);
        atomicAdd(pDst + 1, threadVal10);
        atomicAdd(pDst + 2, threadVal01);
    }
}

template<class InputT, class OutputT, bool nIsPow2>
__global__ void imagePowerMomentsKernel(
        InputT *pSrc, int step,
        OutputT *pDst,
        int width, int height,
        OutputT power) {
    // Shared memory for intermediate steps
    OutputT* sdata = SharedMemory<OutputT>();

    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    // Handle to tile in thread block
    cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cta);
    unsigned int n = step * height;

    unsigned int threadRank = threadIdx.x;
    unsigned int blocksize = cta.size();
    OutputT threadVal00 = 0;
    OutputT threadVal10 = 0;
    OutputT threadVal01 = 0;

    if (nIsPow2) {
        unsigned int i = blockIdx.x * blocksize * 2 + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x << 1;
        while (i < n) {
            imageMomentsPower(
                    pSrc,
                    threadVal00, threadVal10, threadVal01,
                    i,
                    width, height, step,
                    power);
            // ensure we don't read out of bounds -- this is optimized away for
            // powerOf2 sized arrays
            int j = i + blocksize;
            if (j < n) {
                imageMomentsPower(
                        pSrc,
                        threadVal00, threadVal10, threadVal01,
                        j,
                        width, height, step,
                        power);
            }
            i += gridSize;
        }
    }
    else {
        unsigned int i = blockIdx.x * blocksize + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x;
        while (i < n) {
            imageMomentsPower(
                    pSrc,
                    threadVal00, threadVal10, threadVal01,
                    i,
                    width, height, step,
                    power);
            i += gridSize;
        }
    }

    sdata[threadRank] = threadVal00;
    sdata[threadRank + blocksize] = threadVal10;
    sdata[threadRank + 2 * blocksize] = threadVal01;
    cg::sync(cta);

    // do reduction in shared mem
    if (threadRank < 512) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 512];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 512];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 512];
    }
    cg::sync(cta);

    if (threadRank < 256) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 256];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 256];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 256];
    }
    cg::sync(cta);

    if (threadRank < 128) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 128];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 128];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 128];
    }
    cg::sync(cta);

    if (threadRank < 64) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 64];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 64];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 64];
    }
    cg::sync(cta);

    if (threadRank < 32) {
        sdata[threadRank] = threadVal00 = threadVal00 + sdata[threadRank + 32];
        sdata[threadRank + blocksize] = threadVal10 = threadVal10 + sdata[threadRank + blocksize + 32];
        sdata[threadRank + 2 * blocksize] = threadVal01 = threadVal01 + sdata[threadRank + 2 * blocksize + 32];

        // Reduce final warp using shuffle
        for (int offset = tile.size() / 2; offset > 0; offset /= 2) {
            threadVal00 += tile.shfl_down(threadVal00, offset);
            threadVal01 += tile.shfl_down(threadVal01, offset);
            threadVal10 += tile.shfl_down(threadVal10, offset);
        }
    }
    cg::sync(cta);

    if (threadRank == 0) {
        atomicAdd(pDst, threadVal00);
        atomicAdd(pDst + 1, threadVal10);
        atomicAdd(pDst + 2, threadVal01);
    }
}

template<class InputT, class OutputT>
hipError_t imageMomentsImpl(
        InputT *pSrc, int step,
        OutputT *pDst,
        int width, int height,
        OutputT threshold,
        int blocks, int threads,
        hipStream_t& ctx) {

    hipError_t err = hipMemsetAsync(pDst, 0, 3 * sizeof(OutputT), ctx);
    if (err != hipSuccess) return err;

    err = hipStreamSynchronize(ctx);
    if (err != hipSuccess) return err;

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 6 * threads * sizeof(OutputT) : 3 * threads * sizeof(OutputT);

    if (isPow2(height * step)) {
        imageMomentsKernel<InputT, OutputT, true> << <dimGrid, dimBlock, smemSize, ctx >> > (
                pSrc, step, pDst, width, height, threshold);
    }
    else {
        imageMomentsKernel<InputT, OutputT, false> << <dimGrid, dimBlock, smemSize, ctx >> > (
                pSrc, step, pDst, width, height, threshold);
    }


    err = hipGetLastError();
    if (err != hipSuccess) return err;

    err = hipStreamSynchronize(ctx);
    return err;
}

template<class InputT, class OutputT>
hipError_t imagePowerMomentsImpl(
        InputT *pSrc, int step,
        OutputT *pDst,
        int width, int height,
        OutputT power,
        int blocks, int threads,
        hipStream_t& ctx) {

    hipError_t err = hipMemsetAsync(pDst, 0, 3 * sizeof(OutputT), ctx);
    if (err != hipSuccess) return err;

    err = hipStreamSynchronize(ctx);
    if (err != hipSuccess) return err;

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 6 * threads * sizeof(OutputT) : 3 * threads * sizeof(OutputT);
    if (isPow2(height * step)) {
        imagePowerMomentsKernel<InputT, OutputT, true> << <dimGrid, dimBlock, smemSize, ctx >> > (
                pSrc, step, pDst, width, height, power);
    }
    else {
        imagePowerMomentsKernel<InputT, OutputT, false> << <dimGrid, dimBlock, smemSize, ctx >> > (
                pSrc, step, pDst, width, height, power);
    }
    err = hipGetLastError();
    if (err != hipSuccess) return err;

    err = hipStreamSynchronize(ctx);
    return err;
}

hipError_t imageMoments_8u32u(
        unsigned char* pSrc, int step,
        unsigned int *pDst,
        int width, int height,
        unsigned int threshold,
        int blocks, int threads,
        hipStream_t &ctx) {
    return imageMomentsImpl<unsigned char, unsigned int>(
            pSrc, step, pDst, width, height, threshold, blocks, threads, ctx);
}

hipError_t imageMoments_32f(
        float *pSrc, int step,
        float *pDst,
        int width, int height,
        float threshold,
        int blocks, int threads,
        hipStream_t &ctx) {
    return imageMomentsImpl<float, float>(
            pSrc, step, pDst, width, height, threshold, blocks, threads, ctx);
}

hipError_t imageMoments_32f64f(
        float *pSrc, int step,
        double *pDst,
        int width, int height,
        double threshold,
        int blocks, int threads,
        hipStream_t &ctx) {
    return imageMomentsImpl<float, double>(
            pSrc, step, pDst, width, height, threshold, blocks, threads, ctx);

}

hipError_t imagePowerMoments_32f(
        float* pSrc, int step,
        float* pDst,
        int width, int height,
        float power,
        int blocks, int threads,
        hipStream_t& ctx) {
    return imagePowerMomentsImpl<float, float>(
            pSrc, step, pDst, width, height, power, blocks, threads, ctx);
}
