#include "hip/hip_runtime.h"
#include "cu_helper.cuh"
#include "cu_gaussian.cuh"

__global__ void cuGaussianKernel_8u(
		unsigned char *pSrc, int step,
		int width, int height,
		unsigned char amp,
		float sigma) {
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;
	if (tx < width && ty < height)
	{
		float x = tx - (float) width / 2.0f;
		float y = ty - (float) height / 2.0f;
		float r2 = x * x + y * y;
		float val = amp * exp(-r2 / 2.0f / sigma / sigma);
		pSrc[tx + ty * step] = round(val);
	}
}

__global__ void cuGaussianKernel(
		float *pSrc, int step,
		int width, int height,
		float amp,
		float sigma) {
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;
	if (tx < width && ty < height)
	{
		float x = tx - (float) width / 2.0f;
		float y = ty - (float) height / 2.0f;
		float r2 = x * x + y * y;
		pSrc[tx + ty * step] = amp * exp(-r2 / 2.0f / sigma / sigma);
	}
}

void cuGaussain_8u(
		unsigned char *pSrc, int step,
		int width, int height,
		unsigned char amp,
		float sigma,
		hipStream_t& ctx) {

	dim3 dimBlock(32, 32);
	dim3 dimGrid((width - 1 + 32) / 32,
				 (height - 1 + 32) / 32);
	cuGaussianKernel_8u<<<dimGrid, dimBlock, 0, ctx>>>(pSrc, step, width, height, amp, sigma);

	return;
}

void cuGaussain(
		float *pSrc, int step,
		int width, int height,
		float amp,
		float sigma,
		hipStream_t& ctx) {

	dim3 dimBlock(32, 32);
	dim3 dimGrid((width - 1 + 32) / 32,
		         (height - 1 + 32) / 32);
	cuGaussianKernel<<<dimGrid, dimBlock, 0, ctx>>>(pSrc, step, width, height, amp, sigma);
	
	return;
}