#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "cu_helper.cuh"
#include "cu_overlay.cuh"
#include <algorithm>

__global__ void cuOverlayKernel(
        unsigned char *dst, int step,
        int width, int height,
        float xc, float yc, float r,
        unsigned char value) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx > width) | (ty > height)) {
        return;
    }

    float dx = ((float)tx - xc);
    float dy = ((float)ty - yc);
    float d = sqrtf(dx * dx + dy * dy);
    if (d < r) {
        dst[tx + ty * step] = value;
    }
}

__global__ void cuOverlayRectKernel(
        unsigned char *dst, int step,
        int width, int height,
        int x, int y,
        int xend, int yend,
        unsigned char value) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx >= x) & (ty >= y) &
        (tx < xend) & (ty < yend)) {
        dst[tx + ty * step] = value;
    }
}

int cuOverlay(
        unsigned char *dst, int step,
        int width, int height,
        float xc, float yc, float r,
        unsigned char value,
        hipStream_t& ctx) {
	dim3 dimBlock(32, 32);
	dim3 dimGrid((width - 1 + 32) / 32,
				 (height - 1 + 32) / 32);
    cuOverlayKernel<<<dimGrid, dimBlock, 0, ctx>>>(
            dst, step, width, height, xc, yc, r, value);

	return hipSuccess;
}

int cuOverlayRect(
        unsigned char *dst, int step,
        int width, int height,
        int xc, int yc, int w, int h,
        unsigned char value,
        hipStream_t& ctx) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid((width - 1 + 32) / 32,
                 (height - 1 + 32) / 32);

    int x = xc - w/2;
    x = std::max(0, x);
    int xend = x + w;
    xend = std::min(xend, width);
    int y = yc - w/2;
    y = std::max(0, y);
    int yend = y + h;
    xend = std::min(yend, height);

    cuOverlayRectKernel<<<dimGrid, dimBlock, 0, ctx>>>(
            dst, step, width, height, x, y, x + w, y + h, value);

    return hipSuccess;
}

