#include "hip/hip_runtime.h"
#include "cu_helper.cuh"
#include "cu_image_moments.cuh"
#include "cu_radial_symmetry.cuh"

template<typename T, typename InputT, typename OutputT>
__device__ void radialSymmetry(
        InputT* DX, InputT* DY, InputT* DI, InputT* angle, InputT* I,
        OutputT& W, OutputT& BW, OutputT& MW, OutputT& MMW, OutputT& MBW, OutputT& R2,
        int idx,
        int width, int height, int step,
        T cx, T cy,
        int threshold) {
    OutputT x = static_cast<OutputT>(idx % step);
    OutputT y = static_cast<OutputT>(idx / step);
    if ((x < static_cast<OutputT>(width)) & (y < static_cast<OutputT>(height))) {
        OutputT dx = static_cast<OutputT>(DX[idx]);
        OutputT dy = static_cast<OutputT>(DY[idx]);
        OutputT m = 0;
        OutputT b = 0;
        OutputT w = 0;
        OutputT r2 = 0;
        OutputT dI2 = 0;
        OutputT Mag = 0;
        OutputT power2 = static_cast<OutputT>(2);
        if (dx == 0) {
            if (dy == 0) m = static_cast<OutputT>(tan(angle[idx]));
            else m = static_cast<OutputT>(1000000000000);
        }
        else {
            m = dy / dx;
        }

        Mag = static_cast<OutputT>(I[idx]);
        if (Mag > static_cast<OutputT>(threshold)) {
            dI2 = pow(static_cast<OutputT>(DI[idx]), power2);
        }
        else {
            dI2 = 0;
            Mag = 0;
        }

        b = (static_cast<OutputT>(y) - m * static_cast<OutputT>(x));
        r2 = pow(x - static_cast<OutputT>(cx), power2) + pow(y - static_cast<OutputT>(cy), power2);
        w = dI2 / sqrt(r2) / (pow(m, power2) + 1);

        W += w;
        BW += b * w;
        MW += m * w;
        MMW += m * m * w;
        MBW += m * b * w;
        R2 += r2;
    }
}

template<class T, class OutputT, bool nIsPow2>
__global__ void radialSymmetryKernel(
        T* DX,
        T* DY,
        T* DI,
        T* angle,
        T* I,
        int step,
        int width, int height,
        T cx, T cy,
        T threshold,
        OutputT *pDst) {

    // Shared memory for intermediate steps
    OutputT* sdata = SharedMemory<OutputT>();

    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    // Handle to tile in thread block
    cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cta);
    unsigned int n = step * height;
    unsigned int threadRank = threadIdx.x;
    unsigned int blocksize = cta.size();

    OutputT W = 0;
    OutputT BW = 0;
    OutputT MW = 0;
    OutputT MMW = 0;
    OutputT MBW = 0;
    OutputT R2 = 0;

    if (nIsPow2) {
        unsigned int i = blockIdx.x * blocksize * 2 + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x << 1;
        while (i < n) {
            radialSymmetry(
                    DX, DY, DI, angle, I,
                    W, BW, MW, MMW, MBW, R2,
                    i,
                    width, height, step,
                    cx, cy,
                    threshold);

            int j = i + blocksize;
            if (j < n) {
                radialSymmetry(
                        DX, DY, DI, angle, I,
                        W, BW, MW, MMW, MBW, R2,
                        j,
                        width, height, step,
                        cx, cy,
                        threshold);
            }
            i += gridSize;
        }
    }
    else {
        unsigned int i = blockIdx.x * blocksize + threadIdx.x;
        unsigned int gridSize = blocksize * gridDim.x;
        while (i < n) {
            radialSymmetry(
                    DX, DY, DI, angle, I,
                    W, BW, MW, MMW, MBW, R2,
                    i,
                    width, height, step,
                    cx, cy,
                    threshold);
            i += gridSize;
        }
    }

    //reduce(W, sdata, threadRank, cta, tile, 1024);
    //reduce(BW, sdata, threadRank, cta, tile, 1024);
    //reduce(MW, sdata, threadRank, cta, tile, 1024);
    //reduce(MMW, sdata, threadRank, cta, tile, 1024);
    //reduce(MBW, sdata, threadRank, cta, tile, 1024);
    //reduce(R2, sdata, threadRank, cta, tile, 1024);

    sdata[threadRank] = W;
    sdata[threadRank + blocksize] = BW;
    sdata[threadRank + 2 * blocksize] = MW;
    sdata[threadRank + 3 * blocksize] = MMW;
    sdata[threadRank + 4 * blocksize] = MBW;
    sdata[threadRank + 5 * blocksize] = R2;
    cg::sync(cta);

    // do reduction in shared mem
    if ((blocksize >=1024) & (threadRank < 512)) {
        sdata[threadRank] = W = W + sdata[threadRank + 512];
        sdata[threadRank + blocksize] = BW = BW + sdata[threadRank + blocksize + 512];
        sdata[threadRank + 2 * blocksize] = MW = MW + sdata[threadRank + 2 * blocksize + 512];
        sdata[threadRank + 3 * blocksize] = MMW = MMW + sdata[threadRank + 3 * blocksize + 512];
        sdata[threadRank + 4 * blocksize] = MBW = MBW + sdata[threadRank + 4 * blocksize + 512];
        sdata[threadRank + 5 * blocksize] = R2 = R2 + sdata[threadRank + 5 * blocksize + 512];
    }
    cg::sync(cta);

    if ((blocksize >= 512) & (threadRank < 256)) {
        sdata[threadRank] = W = W + sdata[threadRank + 256];
        sdata[threadRank + blocksize] = BW = BW + sdata[threadRank + blocksize + 256];
        sdata[threadRank + 2 * blocksize] = MW = MW + sdata[threadRank + 2 * blocksize + 256];
        sdata[threadRank + 3 * blocksize] = MMW = MMW + sdata[threadRank + 3 * blocksize + 256];
        sdata[threadRank + 4 * blocksize] = MBW = MBW + sdata[threadRank + 4 * blocksize + 256];
        sdata[threadRank + 5 * blocksize] = R2 = R2 + sdata[threadRank + 5 * blocksize + 256];
    }
    cg::sync(cta);

    if ((blocksize >= 256) & (threadRank < 128)) {
        sdata[threadRank] = W = W + sdata[threadRank + 128];
        sdata[threadRank + blocksize] = BW = BW + sdata[threadRank + blocksize + 128];
        sdata[threadRank + 2 * blocksize] = MW = MW + sdata[threadRank + 2 * blocksize + 128];
        sdata[threadRank + 3 * blocksize] = MMW = MMW + sdata[threadRank + 3 * blocksize + 128];
        sdata[threadRank + 4 * blocksize] = MBW = MBW + sdata[threadRank + 4 * blocksize + 128];
        sdata[threadRank + 5 * blocksize] = R2 = R2 + sdata[threadRank + 5 * blocksize + 128];
    }
    cg::sync(cta);

    if ((blocksize >= 128) & (threadRank < 64)) {
        sdata[threadRank] = W = W + sdata[threadRank + 64];
        sdata[threadRank + blocksize] = BW = BW + sdata[threadRank + blocksize + 64];
        sdata[threadRank + 2 * blocksize] = MW = MW + sdata[threadRank + 2 * blocksize + 64];
        sdata[threadRank + 3 * blocksize] = MMW = MMW + sdata[threadRank + 3 *  blocksize + 64];
        sdata[threadRank + 4 * blocksize] = MBW = MBW + sdata[threadRank + 4 * blocksize + 64];
        sdata[threadRank + 5 * blocksize] = R2 = R2 + sdata[threadRank + 5 * blocksize + 64];
    }
    cg::sync(cta);

    if ((blocksize >= 64) & (threadRank < 32)) {
        sdata[threadRank] = W = W + sdata[threadRank + 32];
        sdata[threadRank + blocksize] = BW = BW + sdata[threadRank + blocksize + 32];
        sdata[threadRank + 2 * blocksize] = MW = MW + sdata[threadRank + 2 * blocksize + 32];
        sdata[threadRank + 3 * blocksize] = MMW = MMW + sdata[threadRank + 3 * blocksize + 32];
        sdata[threadRank + 4 * blocksize] = MBW = MBW + sdata[threadRank + 4 * blocksize + 32];
        sdata[threadRank + 5 * blocksize] = R2 = R2 + sdata[threadRank + 5 * blocksize + 32];

        // Reduce final warp using shuffle
        for (int offset = tile.size() / 2; offset > 0; offset /= 2) {
            W += tile.shfl_down(W, offset);
            BW += tile.shfl_down(BW, offset);
            MW += tile.shfl_down(MW, offset);
            MMW += tile.shfl_down(MMW, offset);
            MBW += tile.shfl_down(MBW, offset);
            R2 += tile.shfl_down(R2, offset);
        }
    }
    cg::sync(cta);

    if (threadRank == 0) {
        atomicAdd(pDst, W);
        atomicAdd(pDst + 1, BW);
        atomicAdd(pDst + 2, MW);
        atomicAdd(pDst + 3, MBW);
        atomicAdd(pDst + 4, MMW);
        atomicAdd(pDst + 5, R2);
    }
}

template<class T, class OutputT>
hipError_t radialSymmetryImpl(
        T* DX,
        T* DY,
        T* DI,
        T* angle,
        T* I,
        int step,
        int width, int height,
        T cx, T cy,
        T threshold,
        OutputT *pDst,
        int blocks, int threads,
        hipStream_t& ctx) {
    hipError_t err = hipMemset(pDst, 0, 6 * sizeof(T));
    if (err != hipSuccess) return err;

//    if (threads >= 1024) {
//        blocks = blocks * threads / 512;
//        threads = 512;
//    }

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 12 * threads * sizeof(OutputT) : 6 * threads * sizeof(OutputT);

    if (isPow2(height * step)) {
        radialSymmetryKernel<T, OutputT, true> << <dimGrid, dimBlock, smemSize, ctx >> > (
                DX, DY, DI, angle,
                        I,
                        step,
                        width, height,
                        cx, cy,
                        threshold,
                        pDst);
    }
    else {
        radialSymmetryKernel<T, OutputT, false> << <dimGrid, dimBlock, smemSize, ctx >> > (
                DX, DY, DI, angle,
                        I,
                        step,
                        width, height,
                        cx, cy,
                        threshold,
                        pDst);
    }

    err = hipGetLastError();
    if (err != hipSuccess) return err;

    err = hipStreamSynchronize(ctx);
    return err;
}

hipError_t radialSymmetry_32f(
        float* DX,
        float* DY,
        float* DI,
        float* angle,
        float* I,
        int step,
        int width, int height,
        float cx, float cy,
        float threshold,
        float *pDst,
        int blocks, int threads,
        hipStream_t &ctx) {
    return radialSymmetryImpl<float, float>(
            DX, DY, I, angle,
            I,
            step,
            width, height,
            cx, cy,
            threshold,
            pDst,
            blocks, threads,
            ctx);
}

hipError_t radialSymmetry_64f(
        float* DX,
        float* DY,
        float* DI,
        float* angle,
        float* I,
        int step,
        int width, int height,
        float cx, float cy,
        float threshold,
        double *pDst,
        int blocks, int threads,
        hipStream_t &ctx) {
    return radialSymmetryImpl<float, double>(
            DX, DY, DI, angle,
            I,
            step,
            width, height,
            cx, cy,
            threshold,
            pDst,
            blocks, threads,
            ctx);
}
